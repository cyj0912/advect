#include "hip/hip_runtime.h"
#include "Simulator.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

static vec2 *velocity;
static vec2 *acceleration;
static unsigned int *arrGridHash;
static unsigned int *arrGridParticle;
static unsigned int *gridFirstParticle;

__global__ void initPositions(vec2 *pos)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= particleCount)
        return;
    int i = index / sqrtCount;
    int j = index - i * sqrtCount;
    pos[index] = {boxSideLen / sqrtCount * j + boxOrigin, boxSideLen / sqrtCount * i + boxOrigin};
}

__global__ void integrateOneParticle(vec2 *newPos, vec2 *oldPos, vec2 *v, vec2 *a)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= particleCount)
        return;
    vec2 accel = a[index];
    accel.y -= 9.8;
    v[index] = v[index] + accel * dt;
    newPos[index] = oldPos[index] + v[index] * dt;

    float rdm = (index % 100) / 100.f * radius + radius;
    if (newPos[index].x > 1.0f)
        newPos[index].x = 1.0f - rdm;
    if (newPos[index].x < -1.0f)
        newPos[index].x = -1.0f + rdm;
    if (newPos[index].y > 1.0f)
        newPos[index].y = 1.0f - rdm;
    if (newPos[index].y < -1.0f)
        newPos[index].y = -1.0f + rdm;
}

__global__ void calcGridHash(unsigned int *gridHash, unsigned int *particleIndex, vec2 *newPos)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= particleCount)
        return;
    int2 gridPos;
    gridPos.x = floor((newPos[index].x - boxOrigin) / gridUnitLen);
    gridPos.y = floor((newPos[index].y - boxOrigin) / gridUnitLen);
    gridPos.x = gridPos.x & (gridSideCount - 1);
    gridPos.y = gridPos.y & (gridSideCount - 1);
    unsigned int gh = gridPos.y * gridSideCount + gridPos.x;
    gridHash[index] = gh;
    particleIndex[index] = index;
}

__global__ void findStart(unsigned int *gridStart, unsigned int *gridHash, unsigned int *gridParticle)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= particleCount)
        return;
    if (gridHash[index] != gridHash[index - 1])
        gridStart[gridHash[index]] = index;
    else if (index == 0)
        gridStart[gridHash[0]] = 0;
}

__global__ void doCollision(unsigned int *gridStart, unsigned int *gridHash, unsigned int *gridParticle, vec2 *p,
                            vec2 *v, vec2 *a)
{
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= particleCount)
        return;

    int2 gridPos;
    gridPos.x = floor((p[index].x - boxOrigin) / gridUnitLen);
    gridPos.y = floor((p[index].y - boxOrigin) / gridUnitLen);
    a[index] = {0.0f, 0.0f};
    for (int xOff = -1; xOff <= 1; xOff++)
        for (int yOff = -1; yOff < 1; yOff++)
        {
            int x2 = gridPos.x + xOff;
            int y2 = gridPos.y + yOff;
            x2 = x2 & (gridSideCount - 1);
            y2 = y2 & (gridSideCount - 1);
            unsigned int gh = gridPos.y * gridSideCount + gridPos.x;
            int j = gridStart[gh];
            for (; j < particleCount && gridHash[j] == gh; j++)
            {
                unsigned int jj = gridParticle[j];
                if (jj == index)
                    continue;
                auto force = collidePair(p[index], p[jj], v[index], v[jj], radius, radius, 0.4f);
                a[index] += force / mass;
            }
        }
}

void setup()
{
    hipMallocManaged(&velocity, particleCount * sizeof(vec2));
    hipMallocManaged(&acceleration, particleCount * sizeof(vec2));
    hipMallocManaged(&arrGridHash, particleCount * sizeof(unsigned int));
    hipMallocManaged(&arrGridParticle, particleCount * sizeof(unsigned int));
    hipMallocManaged(&gridFirstParticle, gridSideCount * gridSideCount * sizeof(unsigned int));
}

void teardown()
{
    hipFree(velocity);
    hipFree(acceleration);
    hipFree(arrGridHash);
    hipFree(arrGridParticle);
    hipFree(gridFirstParticle);
}

void simulationStep(vec2 *devNewPos, vec2 *devOldPos)
{
    int blockSz = 1024;
    int blkCnt = (particleCount + blockSz - 1) / blockSz;
    thrust::device_ptr<unsigned int> devGridHash = thrust::device_pointer_cast(arrGridHash);
    thrust::device_ptr<unsigned int> devGridParticle = thrust::device_pointer_cast(arrGridParticle);
    if (!devOldPos)
    {
        setup();
        initPositions<<<blkCnt, blockSz>>>(devNewPos);
        // hipMemset(devNewPos, 0, particleCount * sizeof(vec2));
    }
    else
    {
        integrateOneParticle<<<blkCnt, blockSz>>>(devNewPos, devOldPos, velocity, acceleration);
        calcGridHash<<<blkCnt, blockSz>>>(arrGridHash, arrGridParticle, devNewPos);
        thrust::sort_by_key(devGridHash, devGridHash + particleCount, devGridParticle);
        findStart<<<blkCnt, blockSz>>>(gridFirstParticle, arrGridHash, arrGridParticle);
        doCollision<<<blkCnt, blockSz>>>(gridFirstParticle, arrGridHash, arrGridParticle, devNewPos, velocity,
                                         acceleration);
    }
}
